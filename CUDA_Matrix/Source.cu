#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <windows.h>

using namespace std;

__global__ void matrixMult(const int* A, const int* B, int* C, int size)
{
    int i = size * (blockDim.y * blockIdx.y + threadIdx.y);
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;

    for (int k = 0; k < size; k++)
        sum += A[i + k] * B[k * size + j];

    int ind = size * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[ind] = sum;
}

int main(int argc, char** argv) {
    int threads, size;
    cin >> threads >> size;

    int* A = new int [size * size];
    int* B = new int [size * size];
    int* C = new int [size * size];

    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            A[i * size + j] = B[i * size + j] = i * j;
        }
    }

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int* calcA = NULL;
    hipMalloc((void**)&calcA, size * size);

    int* calcB = NULL;
    hipMalloc((void**)&calcB, size * size);

    int* calcC = NULL;
    hipMalloc((void**)&calcC, size * size);

    hipMemcpy(calcA, A, size * size, hipMemcpyHostToDevice);
    hipMemcpy(calcB, B, size * size, hipMemcpyHostToDevice);

    cout << "Ïðîãðàììà íà÷àëà ñâîþ ðàáîòó" << endl;

    dim3 threadsPerBlock = dim3(threads, threads);
    dim3 blocksPerGrid = dim3(size / threads, size / threads);

    hipEventRecord(start, 0);
    matrixMult <<< blocksPerGrid, threadsPerBlock >>> (calcA, calcB, calcC, size);

    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float kernelTime;
    hipEventElapsedTime(&kernelTime, start, end);

    cout << "Size: " << size << endl;
    cout << "Duration: " << kernelTime / 1000;

    hipFree(calcA);
    hipFree(calcB);
    hipFree(calcC);
    free(A);
    free(B);
    free(C);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
